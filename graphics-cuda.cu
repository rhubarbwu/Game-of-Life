#include "hip/hip_runtime.h"
#include "graphics-cuda.h"

using namespace std;

__host__ uint32_t* gpuAlloc(unsigned screen_size) {
    uint32_t* gpu_mem;

    hipError_t err = hipMalloc(&gpu_mem, screen_size * 4);
    if (err != hipSuccess) return NULL;

    return gpu_mem;
};

__host__ void gpuFree(void* gpu_mem) {
    hipFree(gpu_mem);
}

__host__ unsigned gpuBlit(void* src, void* dst, unsigned screen_size) {
    hipError_t err = hipMemcpy(dst, src, screen_size * 4, hipMemcpyDeviceToHost);
    if (err != hipSuccess) return 1;
    return 0;
}

__device__ uint32_t getPixColor(unsigned* d_field, unsigned S, unsigned H, unsigned W, unsigned x, unsigned y) {
    unsigned i = y / S;
    unsigned j = x / S;
    if (i >= H || j >= W) return 0;

    if (d_field[i * W + j] <= ALIVE)
        return COLOURS[d_field[i * W + j]];
  
    return 0;
}

__global__ void d_render(uint32_t* buf, unsigned* d_field, unsigned S, unsigned H, unsigned W) {
    const unsigned xPix = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned yPix = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned pos = W * S * yPix + xPix;
    buf[pos] = getPixColor(d_field, S, H, W, xPix, yPix);
}

__host__ int render(SDL_Surface *screen,
    void* cuda_pixels,
    unsigned* field, unsigned S, unsigned H, unsigned W) {

    uint32_t*buf = (uint32_t*)cuda_pixels;

    unsigned* d_field;
    hipMalloc((void**)&d_field, (H * W) * sizeof(unsigned));
    hipMemcpy(d_field, field, (H * W) * sizeof(unsigned), hipMemcpyHostToDevice);

    const dim3 blocksPerGrid(W, H);
    const dim3 threadsPerBlock(S, S);
    d_render<<<blocksPerGrid, threadsPerBlock>>>(buf, d_field, S, H, W);
    hipDeviceSynchronize();

    hipMemcpy(field, d_field, (H * W) * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_field);

    return gpuBlit(cuda_pixels, screen->pixels, H*W*S*S);
}
