#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime.h"
#include "field-cuda.h"

using namespace std;

__global__ void d_init(unsigned *d_field, unsigned F) {
    hiprandState state;
    hiprand_init((unsigned long long)clock() + 0, threadIdx.x, 0, &state);
    float rand = hiprand_uniform(&state);
    d_field[threadIdx.x] = rand * MAX < F ? ALIVE : 0;
}

__host__ void init_field(unsigned *field, unsigned H, unsigned W, unsigned F) {
    unsigned *d_field;
    hipMalloc((void **)&d_field, (H * W) * sizeof(unsigned));

    dim3 block_size(H * W);
    dim3 grid_size(1);
    d_init<<<grid_size, block_size>>>(d_field, F);
    hipDeviceSynchronize();

    hipMemcpy(field, d_field, (H * W) * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_field);
}

__global__ void d_transition(unsigned *field, unsigned H, unsigned W) {
    unsigned *prev_field = field;

    unsigned index = threadIdx.x;
    unsigned i = index / W;
    unsigned j = index % W;

    unsigned neighbours = 0;
    NEIGHBOURS

    __syncthreads();
    RULE
}

__host__ void transition(unsigned *field, unsigned H, unsigned W) {
    unsigned *d_field;
    hipMalloc((void **)&d_field, (H * W) * sizeof(unsigned));
    hipMemcpy(d_field, field, (H * W) * sizeof(unsigned), hipMemcpyHostToDevice);

    const dim3 block_size(H * W);
    const dim3 grid_size(1);
    d_transition<<<grid_size, block_size>>>(d_field, H, W);
    hipDeviceSynchronize();

    hipMemcpy(field, d_field, (H * W) * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_field);
}
