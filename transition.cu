#include "hip/hip_runtime.h"
#include "macros.h"

__global__ void transition(int **field, int H, int W) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    bool left = (j > 0 && field[i][j - 1] == ALIVE);
    bool right = (j < W - 1 && field[i][j + 1] == ALIVE);
    bool above = (i > 0 && field[i - 1][j] == ALIVE);
    bool below = (i < H - 1 && field[i + 1][j] == ALIVE);

    int neighbours = left + right + above + below;
    __syncthreads();
    if (i >= H || j >= W) return;
    if (neighbours / 2 == 1) {
        field[i][j] = ALIVE;
    } else {
        field[i][j] = field[i][j] == 0 ? 0 : field[i][j] - 1;
    }
}
