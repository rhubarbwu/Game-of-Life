
#include <hip/hip_runtime.h>
__global__ void transition(int H, int W, bool **prev_field, bool **field) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    bool left = (j > 0 && prev_field[i][j - 1]);
    bool right = (j < W - 1 && prev_field[i][j + 1]);
    bool above = (i > 0 && prev_field[i - 1][j]);
    bool below = (i < H - 1 && prev_field[i + 1][j]);

    int sum = left + right + above + below;
    if (i < H && j < W)
        field[i][j] = sum / 2 == 1;
}
