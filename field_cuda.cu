#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "field_cuda.h"

using namespace std;

__global__ void d_init(unsigned *d_field, unsigned F) {
    hiprandState state;
    hiprand_init((unsigned long long)clock() + 0, threadIdx.x, 0, &state);
    float rand = hiprand_uniform(&state);
    d_field[threadIdx.x] = rand * MAX < F ? ALIVE : 0;
}

__host__ void initField(unsigned *field, unsigned H, unsigned W, unsigned F) {
    unsigned *d_field;
    hipMalloc((void **)&d_field, (H * W) * sizeof(unsigned));

    dim3 block_size(H * W);
    dim3 grid_size(1);
    d_init<<<grid_size, block_size>>>(d_field, F);
    hipDeviceSynchronize();

    hipMemcpy(field, d_field, (H * W) * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_field);
}

__global__ void d_transition(unsigned *d_field, unsigned H, unsigned W) {
    unsigned index = threadIdx.x;
    unsigned i = index / W;
    unsigned j = index % W;

    bool left = (j > 0 && d_field[index - 1] == ALIVE);
    bool right = (j < W - 1 && d_field[index + 1] == ALIVE);
    bool above = (i > 0 && d_field[index - W] == ALIVE);
    bool below = (i < H - 1 && d_field[index + W] == ALIVE);

    unsigned neighbours = left + right + above + below;
    __syncthreads();
    if (neighbours / 2 == 1)
        d_field[index] = ALIVE;
    else
        d_field[index] = d_field[index] == 0 ? 0 : d_field[index] - 1;
}

__host__ void transition(unsigned *field, unsigned H, unsigned W) {
    unsigned *d_field;
    hipMalloc((void **)&d_field, (H * W) * sizeof(unsigned));
    hipMemcpy(d_field, field, (H * W) * sizeof(unsigned), hipMemcpyHostToDevice);

    const dim3 block_size(H * W);
    const dim3 grid_size(1);
    d_transition<<<grid_size, block_size>>>(d_field, H, W);
    hipDeviceSynchronize();

    hipMemcpy(field, d_field, (H * W) * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_field);
}