#include "hip/hip_runtime.h"
#include <stdio.h>

#include "field_cuda.h"

__global__ void d_init(int *d_field, int fill) {
    hiprandState state;
    hiprand_init((unsigned long long)clock() + 0, threadIdx.x, 0, &state);
    float rand = hiprand_uniform(&state);
    d_field[threadIdx.x] = rand * MAX < fill ? ALIVE : 0;
}

__host__ void initField(int *field, int H, int W, int fill) {
    int *d_field;
    hipMalloc((void **)&d_field, (H * W) * sizeof(int));

    dim3 block_size(H * W);
    dim3 grid_size(1);
    d_init<<<grid_size, block_size>>>(d_field, fill);
    hipDeviceSynchronize();

    hipMemcpy(field, d_field, (H * W) * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_field);
}

__global__ void d_transition(int *d_field, int H, int W) {
    int index = threadIdx.x;
    int i = index / W;
    int j = index % W;

    bool left = (j > 0 && d_field[index - 1] == ALIVE);
    if (j == 0) left = false;
    bool right = (j < W - 1 && d_field[index + 1] == ALIVE);
    if (j == W - 1) right = false;
    bool above = (i > 0 && d_field[index - W] == ALIVE);
    if (i == 0) above = false;
    bool below = (i < H - 1 && d_field[index + W] == ALIVE);
    if (i == H - 1) below = false;

    int neighbours = left + right + above + below;
    __syncthreads();
    if (neighbours / 2 == 1)
        d_field[index] = ALIVE;
    else
        d_field[index] = d_field[index] == 0 ? 0 : d_field[index] - 1;
}

__host__ void transition(int *field, int H, int W) {
    int *d_field;
    hipMalloc((void **)&d_field, (H * W) * sizeof(int));

    dim3 block_size(H * W);
    dim3 grid_size(1);

    hipMemcpy(d_field, field, (H * W) * sizeof(int), hipMemcpyHostToDevice);

    d_transition<<<grid_size, block_size>>>(d_field, H, W);
    hipDeviceSynchronize();
    hipMemcpy(field, d_field, (H * W) * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_field);
}