#include <signal.h>

#ifdef _WIN32
#include <Windows.h>
#else
#include <unistd.h>
#endif

#include "helpers.h"
#include "terminal.h"
#include "transition.cu"

int main(int argc, char *argv[]) {
    SANITY;
    ARGUMENTS;

    dim3 block_size(H, W);
    dim3 grid_size(1);

    time_t timer;
    srand((unsigned)time(&timer));

    bool **prev_field, **field;
    hipMallocManaged(&prev_field, H * sizeof(bool *));
    hipMallocManaged(&field, H * sizeof(bool *));
    for (int i = 0; i < H; i++) {
        hipMallocManaged(&prev_field[i], W * sizeof(bool));
        hipMallocManaged(&field[i], W * sizeof(bool));
    }

    for (int i = 0; i < H; i++)
        for (int j = 0; j < W; j++)
            prev_field[i][j] = rand() % 100 < O;

    display(prev_field, H, W);

    struct timespec wait_time = {.tv_sec = REFRESH_S, .tv_nsec = REFRESH_NS};
    int r = 0, t = 0;
    while (!interrupted) {
        nanosleep(&wait_time, NULL);
        if (r++ < R * CLOCK_PER_CYCLE)
            continue;
        else
            r = 0;

        transition<<<grid_size, block_size>>>(H, W, prev_field, field);
        hipDeviceSynchronize();

        if (t++ >= T && T != -1) break;

        clear(H);
        display(field, H, W);

        bool **temp = prev_field;
        prev_field = field;
        field = temp;
    }

    for (int i = 0; i < H; i++) {
        hipFree(prev_field[i]);
        hipFree(field[i]);
    }
    hipFree(prev_field);
    hipFree(field);

    return 0;
}