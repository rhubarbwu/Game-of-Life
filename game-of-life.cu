#include <signal.h>

#ifdef _WIN32
#include <Windows.h>
#else
#include <unistd.h>
#endif

#include "boilerplate.h"
#include "macros.h"
#include "terminal.h"
#include "transition.cu"

int main(int argc, char *argv[]) {
    SANITY;
    ARGUMENTS;

    dim3 block_size(H, W);
    dim3 grid_size(1);

    time_t timer;
    srand((unsigned)time(&timer));

    int **field;
    hipMallocManaged(&field, H * sizeof(int *));
    for (int i = 0; i < H; i++)
        hipMallocManaged(&field[i], W * sizeof(int));

    for (int i = 0; i < H; i++)
        for (int j = 0; j < W; j++)
            field[i][j] = rand() % 100 < O ? ALIVE : 0;

    display(field, H, W);

    struct timespec wait_time = {.tv_sec = REFRESH_S, .tv_nsec = REFRESH_NS};
    int r = 0, t = 0;
    while (!interrupted) {
        nanosleep(&wait_time, NULL);
        if (r++ < R * CLOCK_PER_CYCLE)
            continue;
        else
            r = 0;

        transition<<<grid_size, block_size>>>(field, H, W);
        hipDeviceSynchronize();

        if (t++ >= T && T != -1) break;

        clear(H);
        display(field, H, W);
    }

    for (int i = 0; i < H; i++)
        hipFree(field[i]);
    hipFree(field);

    return 0;
}
